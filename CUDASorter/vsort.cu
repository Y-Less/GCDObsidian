

#include <stdio.h>
#include <stdlib.h>



#include "vsort_kernel.cu"




int main(int argc, char** argv)
{
    int values[NUM];

    for(int i = 0; i < NUM; i++)
    {
        values[i] = rand() % 256;
        printf("%d ", values[i]);
    }
    printf( "\n");
    printf( "------------------------------\n");
    
    

    int * dvalues;
    hipMalloc((void**)&dvalues, sizeof(int) * NUM);
    hipMemcpy(dvalues, values, sizeof(int) * NUM, hipMemcpyHostToDevice);
    
    int * rvalues;
    hipMalloc((void**)&rvalues, sizeof(int) * NUM);
    


    vsort<<<1, NUM, 2* sizeof(int) * NUM>>>((int*)dvalues,(int*)rvalues);

   
    hipMemcpy(values, rvalues, sizeof(int) * NUM, hipMemcpyDeviceToHost);

    hipFree(dvalues);
    hipFree(rvalues);
    

    bool passed = true;
    for(int i = 1; i < NUM; i++)
    {   
        printf( "%d ", values[i-1]); 
        
        if (values[i-1] > values[i])
        {
            passed = false;
            printf("%d ", values[i-1] );
            
        }
    }
    passed ? printf("PASSED :)\n"): printf("FAILED\n");
    

    hipDeviceReset();


}
