
#include <hip/hip_runtime.h>
__global__ void tmerge1(int *input0,int *result0){
  unsigned int tid = threadIdx.x;
  unsigned int bid = blockIdx.x;
  extern __shared__ __attribute__ ((aligned (16))) unsigned char sbase[];
  (( int *)sbase)[tid] = ((tid&256)==0) ? min(input0[((bid*512)+tid)],input0[((bid*512)+(tid^511))]) : max(input0[((bid*512)+tid)],input0[((bid*512)+(tid^511))]);
  __syncthreads();
  (( int *)(sbase + 2048))[tid] = ((tid&128)==0) ? min((( int *)sbase)[tid],(( int *)sbase)[(tid^128)]) : max((( int *)sbase)[tid],(( int *)sbase)[(tid^128)]);
  __syncthreads();
  (( int *)sbase)[tid] = ((tid&64)==0) ? min((( int *)(sbase+2048))[tid],(( int *)(sbase+2048))[(tid^64)]) : max((( int *)(sbase+2048))[tid],(( int *)(sbase+2048))[(tid^64)]);
  __syncthreads();
  (( int *)(sbase + 2048))[tid] = ((tid&32)==0) ? min((( int *)sbase)[tid],(( int *)sbase)[(tid^32)]) : max((( int *)sbase)[tid],(( int *)sbase)[(tid^32)]);
  __syncthreads();
  (( int *)sbase)[tid] = ((tid&16)==0) ? min((( int *)(sbase+2048))[tid],(( int *)(sbase+2048))[(tid^16)]) : max((( int *)(sbase+2048))[tid],(( int *)(sbase+2048))[(tid^16)]);
  __syncthreads();
  (( int *)(sbase + 2048))[tid] = ((tid&8)==0) ? min((( int *)sbase)[tid],(( int *)sbase)[(tid^8)]) : max((( int *)sbase)[tid],(( int *)sbase)[(tid^8)]);
  __syncthreads();
  (( int *)sbase)[tid] = ((tid&4)==0) ? min((( int *)(sbase+2048))[tid],(( int *)(sbase+2048))[(tid^4)]) : max((( int *)(sbase+2048))[tid],(( int *)(sbase+2048))[(tid^4)]);
  __syncthreads();
  (( int *)(sbase + 2048))[tid] = ((tid&2)==0) ? min((( int *)sbase)[tid],(( int *)sbase)[(tid^2)]) : max((( int *)sbase)[tid],(( int *)sbase)[(tid^2)]);
  __syncthreads();
  (( int *)sbase)[tid] = ((tid&1)==0) ? min((( int *)(sbase+2048))[tid],(( int *)(sbase+2048))[(tid^1)]) : max((( int *)(sbase+2048))[tid],(( int *)(sbase+2048))[(tid^1)]);
  __syncthreads();
  result0[((bid*512)+tid)] = (( int *)sbase)[tid];
  
}
