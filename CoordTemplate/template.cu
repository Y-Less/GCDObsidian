
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


// Code Generated with GCDObsidian
__global__ void kernel0(int *input0,int *result0){
  unsigned int tid = threadIdx.x;
  unsigned int bid = blockIdx.x;
  //  extern __shared__ unsigned char sbase[];

  result0[((bid*32)+tid)] =  input0[((bid*32)+tid&15)] + 32;
  
}

// coordination code we want to generate
int coord(int *input0, int input0size, int *output0, int output0size){ 
  
  int* dinput0;
  int* doutput0;

  hipMalloc((void**)&dinput0, sizeof(int) * input0size ); 
  hipMalloc((void**)&doutput0, sizeof(int) * output0size ); 
  hipMemcpy(dinput0, input0, sizeof(int) * input0size, hipMemcpyHostToDevice);
  kernel0<<<1, 32,0 >>>((int*)dinput0,(int*)doutput0);
  hipMemcpy(output0, doutput0, sizeof(int) * 32 , hipMemcpyDeviceToHost);
  hipFree(dinput0);
  hipFree(doutput0);
 
  return 0; // Also. add some error checking... 
}




int main(int argc, char **argv){
  int values[32];
  int result[32];
 

  //generate input data
  for (int i = 0; i < 32; ++i) { 
    values[i] = i; 
  }


  coord(values,32,result,32);
  
  // show results 
  for (int i = 0; i < 32; ++i) { 
    printf("%d ", ((int*)result)[i]);
  }

}

